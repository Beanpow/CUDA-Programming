
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void helloFromGpu()
{
    int i = threadIdx.x;
    int k = threadIdx.y;
    int j = blockIdx.x;
    printf("hello world!%d, %d, %d\n", j, i, k);
}

int main()
{
    const dim3 blockSize(2, 4);
    helloFromGpu<<<2, blockSize>>>();
    hipDeviceSynchronize();
    return 0;
}