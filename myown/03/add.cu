
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;
__global__ void add(const double *x, const double *y, double *z, int N);
void check(const double *z, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(double) * N;
    double *xGPU, *yGPU, *zGPU;

    int blockDimx = 1024;
    int gridDimx = ceil(1.0 *N / blockDimx);
    // printf("%d", N / blockDimx);

    hipMalloc((void**)&xGPU, M);
    hipMalloc((void**)&yGPU, M);
    hipMalloc((void**)&zGPU, M);
    
    double *x = (double*) malloc(M);
    double *y = (double*) malloc(M);
    double *z = (double*) malloc(M);


    for (int n = 0; n < N; ++n)
    {
        x[n] = a;
        y[n] = b;
    }

    hipMemcpy(xGPU, x, M, hipMemcpyHostToDevice);
    hipMemcpy(yGPU, y, M, hipMemcpyHostToDevice);
    hipMemcpy(zGPU, z, M, hipMemcpyHostToDevice);

    add<<<gridDimx, blockDimx>>>(xGPU, yGPU, zGPU, N);

    // cudaDeviceSynchronize();

    hipMemcpy(z, zGPU, M, hipMemcpyDeviceToHost);
    check(z, N);

    free(x);
    free(y);
    free(z);
    return 0;
}

__global__ void add(const double *x, const double *y, double *z, int N)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    // z[i*blockDim.x + j] = x[i*blockDim.x + j] + y[i*blockDim.x + j];
    if(n < N){
        z[n] = x[n] + y[n];
    }
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            printf("%d\n", n);
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

