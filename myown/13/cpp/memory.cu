#include "memory.cuh"
#include <stdlib.h>
#include "error.cuh"

void allocate_memory(int N, int MN, Atom *atom)
{
    atom->NN = (int*) malloc(N * sizeof(int));
    atom->NL = (int*) malloc(N * MN * sizeof(int));
    atom->m  = (real*) malloc(N * sizeof(real));
    atom->x  = (real*) malloc(N * sizeof(real));
    atom->y  = (real*) malloc(N * sizeof(real));
    atom->z  = (real*) malloc(N * sizeof(real));
    atom->vx = (real*) malloc(N * sizeof(real));
    atom->vy = (real*) malloc(N * sizeof(real));
    atom->vz = (real*) malloc(N * sizeof(real));
    atom->fx = (real*) malloc(N * sizeof(real));
    atom->fy = (real*) malloc(N * sizeof(real));
    atom->fz = (real*) malloc(N * sizeof(real));
    atom->pe = (real*) malloc(N * sizeof(real));
    atom->ke = (real*) malloc(N * sizeof(real));
    atom->box = (real*) malloc(6 * sizeof(real));


    CHECK(hipMalloc((void**)&atom->g_NN, N * sizeof(int)));
    CHECK(hipMalloc((void**)&atom->g_NL, N * MN * sizeof(int)));
    CHECK(hipMalloc((void**)&atom->g_x, N * sizeof(real)));
    CHECK(hipMalloc((void**)&atom->g_y, N * sizeof(real)));
    CHECK(hipMalloc((void**)&atom->g_z, N * sizeof(real)));
    CHECK(hipMalloc((void**)&atom->g_fx, N * sizeof(real)));
    CHECK(hipMalloc((void**)&atom->g_fy, N * sizeof(real)));
    CHECK(hipMalloc((void**)&atom->g_fz, N * sizeof(real)));
    CHECK(hipMalloc((void**)&atom->g_pe, N * sizeof(real)));
}

void deallocate_memory(Atom *atom)
{
    free(atom->NN);
    free(atom->NL);
    free(atom->m);
    free(atom->x);
    free(atom->y);
    free(atom->z);
    free(atom->vx);
    free(atom->vy);
    free(atom->vz);
    free(atom->fx);
    free(atom->fy);
    free(atom->fz);
    free(atom->pe);
    free(atom->ke);
    free(atom->box);

    CHECK(hipFree(atom->g_NN));
    CHECK(hipFree(atom->g_NL));
    CHECK(hipFree(atom->g_x));
    CHECK(hipFree(atom->g_y));
    CHECK(hipFree(atom->g_z));
    CHECK(hipFree(atom->g_fx));
    CHECK(hipFree(atom->g_fy));
    CHECK(hipFree(atom->g_fz));
    CHECK(hipFree(atom->g_pe));
}

