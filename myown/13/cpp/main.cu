#include "hip/hip_runtime.h"
#include "common.cuh"
#include "memory.cuh"
#include "initialize.cuh"
#include "neighbor.cuh"
#include "integrate.cuh"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

int main(int argc, char **argv)
{
    int nx = 5;
    int Ne = 20000;
    int Np = 20000;
    hipSetDevice(0);

    if (argc != 3) 
    { 
        printf("Usage: %s nx Ne\n", argv[0]);
        exit(1);
    }
    else
    {
        nx = atoi(argv[1]);
        Ne = atoi(argv[2]);
        Np = Ne;
    }

    int N = 4 * nx * nx * nx;
    int Ns = 100;
    int MN = 200; // max number of neighbors
    real T_0 = 60.0;
    real ax = 5.385;
    real time_step = 5.0 / TIME_UNIT_CONVERSION;
    Atom atom;
    allocate_memory(N, MN, &atom);
    for (int n = 0; n < N; ++n) { atom.m[n] = 40.0; }
    initialize_position(nx, ax, &atom);
    initialize_velocity(N, T_0, &atom);
    find_neighbor(N, MN, &atom);
    clock_t startTime = clock();
    CHECK(hipDeviceSynchronize());
    equilibration(Ne, N, MN, T_0, time_step, &atom);
    printf("%g\n", float(clock() - startTime) / CLOCKS_PER_SEC);
    production(Np, Ns, N, MN, T_0, time_step, &atom);
    printf("%g\n", float(clock() - startTime) / CLOCKS_PER_SEC);
    deallocate_memory(&atom);
    return 0;
}

