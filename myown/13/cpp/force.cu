#include "hip/hip_runtime.h"
#include "force.cuh"
#include "mic.cuh"
#include "error.cuh"

struct Constants
{
    real cutoff_square;
    real e24s6;
    real e48s12;
    real e4s6;
    real e4s12;
    int MaxN;
};

void __global__ gpu_find_force(
    Constants constants, int N, int *g_NN, int *g_NL, real *box,
    real *g_x, real *g_y, real *g_z, real *g_pe, real *g_fx, real *g_fy, real *g_fz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
        return;

    for (int k = 0; k < g_NN[i]; k++)
    {
        // int j = g_NL[i * constants.MaxN + k];
        continue;
        // if (j < i)
        //     continue;

        // real dx = g_x[i] - g_x[j];
        // real dy = g_y[i] - g_y[j];
        // real dz = g_z[i] - g_z[j];
        // apply_mic(box, &dx, &dy, &dz);

        // real r2 = dx * dx + dy * dy + dz * dz;
        // if (r2 > constants.cutoff_square)
        //     continue;

        // real r2inv = 1.0 / r2;
        // real r4inv = r2inv * r2inv;
        // real r6inv = r2inv * r4inv;
        // real r8inv = r4inv * r4inv;
        // real r12inv = r4inv * r8inv;
        // real r14inv = r6inv * r8inv;

        // real f = constants.e24s6 * r8inv - constants.e48s12 * r14inv;
        // g_pe[i] += constants.e4s12 * r12inv - constants.e4s12 * r14inv;
        // g_fx[i] += f * dx;
        // g_fx[j] -= f * dx;
        // g_fy[i] += f * dy;
        // g_fy[j] -= f * dy;
        // g_fz[i] += f * dz;
        // g_fz[j] -= f * dz;
    }
}

void find_force(int N, int MN, Atom *atom)
{
    int *NN = atom->NN;
    int *NL = atom->NL;
    real *x = atom->x;
    real *y = atom->y;
    real *z = atom->z;
    real *fx = atom->fx;
    real *fy = atom->fy;
    real *fz = atom->fz;
    real *pe = atom->pe;
    real *box = atom->box;
    const real epsilon = 1.032e-2;
    const real sigma = 3.405;
    const real cutoff = 10.0;
    const real cutoff_square = cutoff * cutoff;
    const real sigma_3 = sigma * sigma * sigma;
    const real sigma_6 = sigma_3 * sigma_3;
    const real sigma_12 = sigma_6 * sigma_6;
    const real e24s6 = 24.0 * epsilon * sigma_6;
    const real e48s12 = 48.0 * epsilon * sigma_12;
    const real e4s6 = 4.0 * epsilon * sigma_6;
    const real e4s12 = 4.0 * epsilon * sigma_12;

    Constants constants;
    constants.cutoff_square = cutoff_square;
    constants.e24s6 = e24s6;
    constants.e48s12 = e48s12;
    constants.e4s12 = e4s12;
    constants.e4s6 = e4s6;
    constants.MaxN = MN;

    int blockSize = 128;
    int gridSize = (N - 1) / blockSize + 1;
    int m = sizeof(real) * N;

    for (int n = 0; n < N; ++n)
    {
        fx[n] = fy[n] = fz[n] = pe[n] = 0.0;
    }

    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(atom->g_NL, atom->NL, N * MN * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_NN, NN, N * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_x, atom->x, m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_y, atom->y, m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_z, atom->z, m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_pe, atom->pe, m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_fx, atom->fx, m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_fy, atom->fy, m, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(atom->g_fz, atom->fz, m, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());


    gpu_find_force<<<gridSize, blockSize>>>(
        constants, N, atom->g_NN, atom->g_NL, box, atom->g_x, atom->g_y, atom->g_z,
        atom->pe, atom->fx, atom->fy, atom->fz);
}
